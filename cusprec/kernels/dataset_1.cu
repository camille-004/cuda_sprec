
#include <hip/hip_runtime.h>
__global__ void initialize(
    float *x,
    float *A,
    float *b,
    float *rand_vals,
    int *rand_indices,
    int m,
    int n,
    int s
) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < s) {
        int i = rand_indices[idx];
        x[i] = rand_vals[i * (m + 1)];
    }

    if (idx < n) {
        for (int i = 0; i < m; i++) {
            A[i * n + idx] = rand_vals[idx + i + 1];
        }

        float sum_squares = 0.0f;
        for (int i = 0; i < m; i++) {
            sum_squares += A[i * n + idx] * A[i * n + idx];
        }
        float norm = sqrtf(sum_squares);

        for (int i = 0; i < m; i++) {
            A[i * n + idx] /= norm;
        }
    }

    if (idx < m) {
        float sum = 0.0f;
        for (int j = 0; j < n; j++) {
            sum += A[idx * n + j] * x[j];
        }
        b[idx] = sum;
    }
}